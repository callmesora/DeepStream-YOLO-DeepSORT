/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */

#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <string.h>

__global__ void gpuYoloLayer_nc(
    const float* input, float* output, const uint netWidth, const uint netHeight, const uint gridSizeX,
    const uint gridSizeY, const uint numOutputClasses, const uint numBBoxes, const float scaleXY,
    const float* anchors, const int* mask)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if ((x_id >= gridSizeX) || (y_id >= gridSizeY) || (z_id >= numBBoxes))
    {
        return;
    }

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    const float alpha = scaleXY;
    const float beta = -0.5 * (scaleXY - 1);

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
        = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
          * alpha + beta + x_id) * netWidth / gridSizeX;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
        = (input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
          * alpha + beta + y_id) * netHeight / gridSizeY;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]
        = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)] * 2, 2)
          * anchors[mask[z_id] * 2];

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]
        = __powf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)] * 2, 2)
          * anchors[mask[z_id] * 2 + 1];

    const float objectness
        = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)];

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]
        = objectness * maxProb;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 5)]
        = maxIndex;
}

hipError_t cudaYoloLayer_nc(
    const void* input, void* output, const uint& batchSize, const uint& netWidth, const uint& netHeight,
    const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    uint64_t& outputSize, const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream);

hipError_t cudaYoloLayer_nc(
    const void* input, void* output, const uint& batchSize, const uint& netWidth, const uint& netHeight,
    const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses, const uint& numBBoxes,
    uint64_t& outputSize, const float& scaleXY, const void* anchors, const void* mask, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuYoloLayer_nc<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * outputSize),
            reinterpret_cast<float*>(output) + (batch * outputSize),
            netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes, scaleXY,
            reinterpret_cast<const float*>(anchors), reinterpret_cast<const int*>(mask));
    }
    return hipGetLastError();
}
