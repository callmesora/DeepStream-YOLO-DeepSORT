/*
 * Created by Marcos Luciano
 * https://www.github.com/marcoslucianops
 */

#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <string.h>

inline __device__ float sigmoidGPU(const float& x) { return 1.0f / (1.0f + __expf(-x)); }

__device__ void softmaxGPU(
    const float* input, const int bbindex, const int numGridCells, uint z_id,
    const uint numOutputClasses, float temp, float* output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for (i = 0; i < numOutputClasses; ++i) {
        int val = input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];
        largest = (val>largest) ? val : largest;
    }
    for (i = 0; i < numOutputClasses; ++i) {
        float e = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] / temp - largest / temp);
        sum += e;
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] = e;
    }
    for (i = 0; i < numOutputClasses; ++i) {
        output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))] /= sum;
    }
}

__global__ void gpuRegionLayer(
    const float* input, float* output, float* softmax, const uint netWidth, const uint netHeight,
    const uint gridSizeX, const uint gridSizeY, const uint numOutputClasses, const uint numBBoxes,
    const float* anchors)
{
    uint x_id = blockIdx.x * blockDim.x + threadIdx.x;
    uint y_id = blockIdx.y * blockDim.y + threadIdx.y;
    uint z_id = blockIdx.z * blockDim.z + threadIdx.z;

    if ((x_id >= gridSizeX) || (y_id >= gridSizeY) || (z_id >= numBBoxes))
    {
        return;
    }

    const int numGridCells = gridSizeX * gridSizeY;
    const int bbindex = y_id * gridSizeX + x_id;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)]
        = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 0)])
          + x_id) * netWidth / gridSizeX;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)]
        = (sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 1)])
          + y_id) * netHeight / gridSizeY;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)]
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 2)])
          * anchors[z_id * 2] * netWidth / gridSizeX;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)]
        = __expf(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 3)])
          * anchors[z_id * 2 + 1] * netHeight / gridSizeY;

    softmaxGPU(input, bbindex, numGridCells, z_id, numOutputClasses, 1.0, softmax);

    const float objectness
        = sigmoidGPU(input[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]);

    float maxProb = 0.0f;
    int maxIndex = -1;

    for (uint i = 0; i < numOutputClasses; ++i)
    {
        float prob
            = softmax[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + (5 + i))];

        if (prob > maxProb)
        {
            maxProb = prob;
            maxIndex = i;
        }
    }

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 4)]
        = objectness * maxProb;

    output[bbindex + numGridCells * (z_id * (5 + numOutputClasses) + 5)]
        = maxIndex;
}

hipError_t cudaRegionLayer(
    const void* input, void* output, void* softmax, const uint& batchSize, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses,
    const uint& numBBoxes, uint64_t& outputSize, const void* anchors, hipStream_t stream);

hipError_t cudaRegionLayer(
    const void* input, void* output, void* softmax, const uint& batchSize, const uint& netWidth,
    const uint& netHeight, const uint& gridSizeX, const uint& gridSizeY, const uint& numOutputClasses,
    const uint& numBBoxes, uint64_t& outputSize, const void* anchors, hipStream_t stream)
{
    dim3 threads_per_block(16, 16, 4);
    dim3 number_of_blocks((gridSizeX / threads_per_block.x) + 1,
                          (gridSizeY / threads_per_block.y) + 1,
                          (numBBoxes / threads_per_block.z) + 1);

    for (unsigned int batch = 0; batch < batchSize; ++batch)
    {
        gpuRegionLayer<<<number_of_blocks, threads_per_block, 0, stream>>>(
            reinterpret_cast<const float*>(input) + (batch * outputSize),
            reinterpret_cast<float*>(output) + (batch * outputSize),
            reinterpret_cast<float*>(softmax) + (batch * outputSize),
            netWidth, netHeight, gridSizeX, gridSizeY, numOutputClasses, numBBoxes,
            reinterpret_cast<const float*>(anchors));
    }
    return hipGetLastError();
}
